#include "hip/hip_runtime.h"
/* -*- mode: c -*- */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <slang.h>
#include "slcuda.h"

SLANG_MODULE(cudaimage);

__global__ void _cuda_image_smooth (float *img, float *kernel, float *out,
				    int kx, int ky, int stride, int N)
{
  int x=THREADIDX;
  if (x>N)
    return;

  int imgx=x%stride;
  int imgy=x/stride;
  int idxx,idxy;
  out[x]=0;
  // loop over all items in kernel
  for (int i=0;i<kx;i++){
    for (int j=0;j<ky;j++){
      // need to mirror if on edge
      idxx=abs(imgx+(i-kx/2));
      idxy=abs(imgy+(j-ky/2));
      if (idxx >= stride)   idxx=2*stride-idxx-1;
      if (idxy >= N/stride) idxy=2*N/stride-idxy-1;
      out[x]+=kernel[j*kx+i]*img[idxy*stride+idxx];
    }
  }
}

__global__ void _cuda_vector_smooth (float *img, float *kernel, float *out,
				    int kx, int N)
{
  int x=THREADIDX;
  if (x>N)
    return;

  int idx;
  out[x]=0;
  // loop over all items in kernel
  for (int i=0;i<kx;i++){
      // need to mirror if on edge
      idx=abs(x+(i-kx/2));
      if (idx >= N) idx=2*N-idx-1;
      out[x]+=kernel[i]*img[idx];
  }
}

static void slcuda_smooth (void)
{
  SLcuda_Type *cuda_img;
  SLcuda_Type *cuda_kernel;
  SLcuda_Type *cuda_o;

  // if we are given three args, then the output goes to that cuda
  // object
  if (3==SLang_Num_Function_Args)
    if (NULL==(cuda_o=slcuda_pop_cuda()))
      return;
  // get image and kernel
  if (NULL==(cuda_kernel=slcuda_pop_cuda()))
    return;
  if (NULL==(cuda_img=slcuda_pop_cuda()))
    return;

  // Image and kernel should be no more than 2d and kernel needs to be
  // odd by odd dimensions
  if (0==cuda_kernel->dims[0]%2||
      2<cuda_kernel->ndims||
      2<cuda_img->ndims){
    printf("Wrong kernel/image dimensions for smoothing\n");
    return;
  }
  if (2==cuda_kernel->ndims&&
      0==cuda_kernel->dims[1]%2){
    printf("kernel 2nd dimension must be odd\n");
    return;
  }
  // if we are not given a device array to fill, make new one to
  // return
  if (3!=SLang_Num_Function_Args){
    cuda_o=slcuda_init_cuda(cuda_img->size,
			    SLANG_FLOAT_TYPE,
			    cuda_img->ndims,
			    cuda_img->dims);
  }
  
  int dx, dy;
  slcuda_compute_dims2d(cuda_img->nelems, SLCUDA_BLOCK_SIZE, &dx, &dy);
  dim3 n_blocks(dx, dy);

  // should handle 1 or 2 dimensions here
  if (2==cuda_img->ndims)
    _cuda_image_smooth <<<
      n_blocks, SLCUDA_BLOCK_SIZE
		       >>> ((float *)cuda_img->dptr,
			    (float *)cuda_kernel->dptr,
			    (float *)cuda_o->dptr,
			    cuda_kernel->dims[0],
			    cuda_kernel->dims[1],
			    cuda_img->dims[1],
			    cuda_img->nelems);
  else
    _cuda_vector_smooth <<<
      n_blocks, SLCUDA_BLOCK_SIZE
			>>> ((float *)cuda_img->dptr,
			     (float *)cuda_kernel->dptr,
			     (float *)cuda_o->dptr,
			     cuda_kernel->dims[0],
			     cuda_img->nelems);
    

  if (3!=SLang_Num_Function_Args)
    slcuda_push_cuda(cuda_o);
}

// forward declaration, def below
static void _test_smooth (void);

static SLang_Intrin_Fun_Type Module_Intrinsics [] =
{
  MAKE_INTRINSIC_0("cusmooth", slcuda_smooth, SLANG_VOID_TYPE),
  MAKE_INTRINSIC_0("testsmooth", _test_smooth, SLANG_VOID_TYPE),
  SLANG_END_INTRIN_FUN_TABLE
};

  
int init_cudaimage_module_ns (char *ns_name)
{
   SLang_NameSpace_Type *ns;

   if (NULL == (ns = SLns_create_namespace (ns_name)))
     return -1;

   if ((-1 == SLns_add_intrin_fun_table (ns, Module_Intrinsics, NULL))
       )
     return -1;
   
   return 0;
}

void deinit_cudaimage_module (void)
{
}

// FOR TEST PURPOSES!
// non-device image smoothing for comparison
static void _test_smooth (void){
  SLang_Array_Type *arr;
  SLang_Array_Type *ker;
  SLang_Array_Type *aro;
  if (-1==SLang_pop_array(&ker,0)) return;
  if (-1==SLang_pop_array(&arr,0)) return;
  int N = arr->num_elements;
  int stride = arr->dims[0];
  int kx = ker->dims[0];
  int ky = ker->dims[1];
  int i,j,x,ix,iy,idxx,idxy;
  float *img = (float *)arr->data;
  float *kernel = (float *)ker->data;
  float *out;
  out = (float *)malloc(N*sizeof(float));
  for (x=0;x<N;x++){
    ix=x%stride;
    iy=x/stride;
    out[x]=0;
    for (i=0;i<kx;i++){
      for (j=0;j<ky;j++){
	// need to mirror if on edge
	idxx=abs(ix+(i-kx/2));
	idxy=abs(iy+(j-ky/2));
	if (idxx >= stride)   idxx=2*stride-idxx-1;
	if (idxy >= N/stride) idxy=2*N/stride-idxy-1;
	out[x]+=kernel[j*kx+i]*img[idxy*stride+idxx];
      }
    }
  }
  aro = SLang_create_array(SLANG_FLOAT_TYPE, 0, (VOID_STAR)out,
			   arr->dims, arr->num_dims);
  SLang_push_array(aro,1);
}
